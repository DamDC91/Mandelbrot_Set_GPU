#include "hip/hip_runtime.h"
#include <view.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <string>

void move(view *v, double dx, double dy)
{
    v->Xmax+=dx;
    v->Xmin+=dx;
    v->Ymax+=dy;
    v->Ymin+=dy;
}

void zoom(view *v, double d)
{
    double dY= std::abs(v->Ymax-v->Ymin)*d;
    double dX= std::abs(v->Xmax-v->Xmin)*d;
    v->Ymax-=dY;
    v->Ymin+=dY;
    v->Xmax-=dX;
    v->Xmin+=dX;
}

void save(view *v, std::string fileName)
{
    std::ofstream wf(fileName, std::ofstream::out | std::ofstream::binary);
    if(!wf) 
    {
      std::cerr << "Cannot open file!" << std::endl;
      return;
    }
    wf.write((char *)v , sizeof(*v));
    wf.close();
    std::cout << "view saved" << std::endl;
}

view *load(std::string fileName)
{
    std::ifstream rf(fileName, std::ofstream::in | std::ifstream::binary);
    if(!rf) 
    {
      std::cerr << "Cannot open file!" << std::endl;
      return nullptr;
    }
    rf.seekg (0, rf.end);
    int length = rf.tellg();
    if (length != sizeof(view))
        std::cerr << "invalid file " << fileName << std::endl;
    rf.seekg (0, rf.beg);
    char *buffer = new char[length];
    rf.read(buffer , length);
    rf.close();
    std::cout << "view loaded" << std::endl;
    return (view *) buffer;
}

void capture(const unsigned char *pixels, int windowSizeX, int windowSizeY, std::string fileName)
{
    std::ofstream wf(fileName, std::ofstream::out | std::ofstream::binary);
    if(!wf) 
    {
      std::cerr << "Cannot open file!" << std::endl;
      return;
    }
    wf << "P6" << std::endl << windowSizeX << ' ' << windowSizeY << std::endl << "255" << std::endl;
 
    for (auto i = 0; i < windowSizeX * windowSizeY * 4; i+=4)
        wf << pixels[i] << pixels[i+1] << pixels[i+2];

    wf.close();
    std::cout << "captured" << std::endl;
}